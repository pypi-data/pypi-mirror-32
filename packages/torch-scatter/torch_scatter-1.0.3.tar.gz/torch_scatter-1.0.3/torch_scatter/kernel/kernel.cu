#include "hip/hip_runtime.h"
#include <THC.h>

#include "kernel.h"

#include "common.cuh"
#include "THCIndex.cuh"
#include "THCAtomics.cuh"

#define scatter_(NAME) TH_CONCAT_4(scatter_, NAME, _kernel_, Real)
#define index_backward TH_CONCAT_2(index_backward_kernel_, Real)
#define thc_(NAME) TH_CONCAT_4(thc_, NAME, _, Real)

#include "generic/common.cu"
#include "THCGenerateAllTypes.h"

template<typename Real, int Dims>
__global__ void mulKernel(TensorInfo<Real> output, TensorInfo<int64_t> index, TensorInfo<Real> input, const int dim, const int n) {
  KERNEL_LOOP(i, n) {
    int outputOffset = 0; int indexOffset = 0; int inputOffset = 0;
    IndexToScatterOffsets3<Real, Real, Dims>::compute(i, dim, index, &indexOffset, input, &inputOffset, output, &outputOffset);
    atomMul(&output.data[outputOffset], input.data[inputOffset]);
  }
}

template<typename Real, int Dims>
__global__ void divKernel(TensorInfo<Real> output, TensorInfo<int64_t> index, TensorInfo<Real> input, const int dim, const int n) {
  KERNEL_LOOP(i, n) {
    int outputOffset = 0; int indexOffset = 0; int inputOffset = 0;
    IndexToScatterOffsets3<Real, Real, Dims>::compute(i, dim, index, &indexOffset, input, &inputOffset, output, &outputOffset);
    atomDiv(&output.data[outputOffset], input.data[inputOffset]);
  }
}

template<typename Real, int Dims>
__global__ void meanKernel(TensorInfo<Real> output, TensorInfo<int64_t> index, TensorInfo<Real> input, TensorInfo<Real> count, const int dim, const int n) {
  KERNEL_LOOP(i, n) {
    int outputOffset = 0; int indexOffset = 0; int inputOffset = 0; int countOffset = 0;
    IndexToScatterOffsets4<Real, Real, Real, Dims>::compute(i, dim, index, &indexOffset, input, &inputOffset, output, &outputOffset, count, &countOffset);
    atomAdd(&output.data[outputOffset], input.data[inputOffset]);
    atomAdd(&count.data[countOffset], 1);
  }
}

template<typename Real, int Dims>
__global__ void maxKernel(TensorInfo<Real> output, TensorInfo<int64_t> index, TensorInfo<Real> input, const int dim, const int n) {
  KERNEL_LOOP(i, n) {
    int outputOffset = 0; int indexOffset = 0; int inputOffset = 0;
    IndexToScatterOffsets3<Real, Real, Dims>::compute(i, dim, index, &indexOffset, input, &inputOffset, output, &outputOffset);
    atomMax(&output.data[outputOffset], input.data[inputOffset]);
  }
}

template<typename Real, int Dims>
__global__ void minKernel(TensorInfo<Real> output, TensorInfo<int64_t> index, TensorInfo<Real> input, const int dim, const int n) {
  KERNEL_LOOP(i, n) {
    int outputOffset = 0; int indexOffset = 0; int inputOffset = 0;
    IndexToScatterOffsets3<Real, Real, Dims>::compute(i, dim, index, &indexOffset, input, &inputOffset, output, &outputOffset);
    atomMin(&output.data[outputOffset], input.data[inputOffset]);
  }
}

template<typename Real, int Dims>
__global__ void argKernel(TensorInfo<Real> output, TensorInfo<int64_t> index, TensorInfo<Real> input, TensorInfo<int64_t> arg, const int dim, const int n) {
  KERNEL_LOOP(i, n) {
    int outputOffset = 0; int indexOffset = 0; int inputOffset = 0; int argOffset = 0;
    IndexToScatterOffsets4<Real, Real, int64_t, Dims>::compute(i, dim, index, &indexOffset, input, &inputOffset, output, &outputOffset, arg, &argOffset);
    if (input.data[inputOffset] == output.data[outputOffset]) {
      arg.data[argOffset] = (inputOffset / input.stride[dim]) % input.size[dim];
    }
  }
}

template<typename Real, int Dims>
__global__ void indexBackwardKernel(TensorInfo<Real> output, TensorInfo<int64_t> index, TensorInfo<Real> grad, TensorInfo<int64_t> arg, const int dim, const int n) {
  KERNEL_LOOP(i, n) {
    int outputOffset = 0; int indexOffset = 0; int gradOffset = 0; int argOffset = 0;
    IndexToScatterOffsets4<Real, Real, int64_t, Dims>::compute(i, dim, index, &indexOffset, output, &outputOffset, grad, &gradOffset, arg, &argOffset);
    if (arg.data[argOffset] == (outputOffset / output.stride[dim]) % output.size[dim]) {
      output.data[outputOffset] = grad.data[gradOffset];
    }
  }
}

#include "generic/kernel.cu"
#include "THCGenerateFloatType.h"
#include "generic/kernel.cu"
#include "THCGenerateDoubleType.h"
#include "generic/kernel.cu"
#include "THCGenerateByteType.h"
#include "generic/kernel.cu"
#include "THCGenerateCharType.h"
#include "generic/kernel.cu"
#include "THCGenerateShortType.h"
#include "generic/kernel.cu"
#include "THCGenerateIntType.h"
#include "generic/kernel.cu"
#include "THCGenerateLongType.h"
