
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

void scatter_(mul)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input) {
  thc_(check)(state, output, index, input);

  const int n = THCudaLongTensor_nElement(state, index);
  TensorInfo<real> outputInfo = thc_(getTensorInfo)(state, output);
  TensorInfo<int64_t> indexInfo = thc_getTensorInfo_Long(state, index);
  TensorInfo<real> inputInfo = thc_(getTensorInfo)(state, input);

  KERNEL_RUN(mulKernel, indexInfo.dims, n, outputInfo, indexInfo, inputInfo, dim)
}

void scatter_(div)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input) {
  thc_(check)(state, output, index, input);

  const int n = THCudaLongTensor_nElement(state, index);
  TensorInfo<real> outputInfo = thc_(getTensorInfo)(state, output);
  TensorInfo<int64_t> indexInfo = thc_getTensorInfo_Long(state, index);
  TensorInfo<real> inputInfo = thc_(getTensorInfo)(state, input);

  KERNEL_RUN(divKernel, indexInfo.dims, n, outputInfo, indexInfo, inputInfo, dim)
}

void scatter_(mean)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCTensor *count) {
  thc_(check)(state, output, index, input);

  const int n = THCudaLongTensor_nElement(state, index);
  TensorInfo<real> outputInfo = thc_(getTensorInfo)(state, output);
  TensorInfo<int64_t> indexInfo = thc_getTensorInfo_Long(state, index);
  TensorInfo<real> inputInfo = thc_(getTensorInfo)(state, input);
  TensorInfo<real> countInfo = thc_(getTensorInfo)(state, count);

  KERNEL_RUN(meanKernel, indexInfo.dims, n, outputInfo, indexInfo, inputInfo, countInfo, dim)
}

void scatter_(max)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCudaLongTensor *arg) {
  thc_(check)(state, output, index, input);

  const int n = THCudaLongTensor_nElement(state, index);
  TensorInfo<real> outputInfo = thc_(getTensorInfo)(state, output);
  TensorInfo<int64_t> indexInfo = thc_getTensorInfo_Long(state, index);
  TensorInfo<real> inputInfo = thc_(getTensorInfo)(state, input);
  TensorInfo<int64_t> argInfo = thc_getTensorInfo_Long(state, arg);

  KERNEL_RUN(maxKernel, indexInfo.dims, n, outputInfo, indexInfo, inputInfo, dim)
  KERNEL_RUN(argKernel, indexInfo.dims, n, outputInfo, indexInfo, inputInfo, argInfo, dim)
}

void scatter_(min)(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *input, THCudaLongTensor *arg) {
  thc_(check)(state, output, index, input);

  const int n = THCudaLongTensor_nElement(state, index);
  TensorInfo<real> outputInfo = thc_(getTensorInfo)(state, output);
  TensorInfo<int64_t> indexInfo = thc_getTensorInfo_Long(state, index);
  TensorInfo<real> inputInfo = thc_(getTensorInfo)(state, input);
  TensorInfo<int64_t> argInfo = thc_getTensorInfo_Long(state, arg);

  KERNEL_RUN(minKernel, indexInfo.dims, n, outputInfo, indexInfo, inputInfo, dim)
  KERNEL_RUN(argKernel, indexInfo.dims, n, outputInfo, indexInfo, inputInfo, argInfo, dim)
}

void index_backward(THCState *state, int dim, THCTensor *output, THCudaLongTensor *index, THCTensor *grad, THCudaLongTensor *arg) {
  thc_(check)(state, output, index, grad);

  const int n = THCudaLongTensor_nElement(state, index);
  TensorInfo<real> outputInfo = thc_(getTensorInfo)(state, output);
  TensorInfo<int64_t> indexInfo = thc_getTensorInfo_Long(state, index);
  TensorInfo<real> gradInfo = thc_(getTensorInfo)(state, grad);
  TensorInfo<int64_t> argInfo = thc_getTensorInfo_Long(state, arg);

  KERNEL_RUN(indexBackwardKernel, indexInfo.dims, n, outputInfo, indexInfo, gradInfo, argInfo, dim)
}

#endif
