#include "hip/hip_runtime.h"
#include "THCWeighting.h"

#include "common.cuh"
#include "THCNumerics.cuh"
#include "THCAtomics.cuh"

#define TH_TENSOR_WEIGHTING(NAME, N, TENSOR1, TENSOR2, TENSOR3, TENSOR4, weightIndex) { \
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 5, TENSOR1, TENSOR2, TENSOR3, TENSOR4, weightIndex)); \
\
  TensorInfo<real> tensor1Info = THCTensor_(getTensorInfo)(state, TENSOR1); \
  TensorInfo<real> tensor2Info = THCTensor_(getTensorInfo)(state, TENSOR2); \
  TensorInfo<real> tensor3Info = THCTensor_(getTensorInfo)(state, TENSOR3); \
  TensorInfo<real> tensor4Info = THCTensor_(getTensorInfo)(state, TENSOR4); \
  TensorInfo<int64_t> weightIndexInfo = THCudaLongTensor_getTensorInfo(state, weightIndex); \
\
  KERNEL_REAL_RUN(NAME, N, tensor1Info, tensor2Info, tensor3Info, tensor4Info, weightIndexInfo); \
}

template<typename T>
__global__ void weightingForwardKernel(TensorInfo<T> self, TensorInfo<T> src, TensorInfo<T> weight,
                                       TensorInfo<T> basis, TensorInfo<int64_t> weightIndex,
                                       int n) {
  KERNEL_LOOP(i, n) {
    ptrdiff_t e = i / self.size[1], mOut = i % self.size[1], s, mIn;
    T v = ScalarConvert<int, T>::to(0), b, tmp;
    int64_t wi;
    for (s = 0; s < basis.size[1]; s++) {
      b = basis.data[e * basis.stride[0] + s * basis.stride[1]];
      wi = weightIndex.data[e * weightIndex.stride[0] + s * weightIndex.stride[1]];
      for (mIn = 0; mIn < src.size[1]; mIn++) {
        tmp = weight.data[wi * weight.stride[0] + mIn * weight.stride[1] + mOut * weight.stride[2]];
        tmp = THCNumerics<T>::mul(tmp, src.data[e * src.stride[0] + mIn * src.stride[1]]);
        tmp = THCNumerics<T>::mul(tmp, b);
        v = THCNumerics<T>::add(v, tmp);
      }
    }
    self.data[e * self.stride[0] + mOut * self.stride[1]] = v;
  }
}

template<typename T>
__global__ void weightingBackwardSrcKernel(TensorInfo<T> self, TensorInfo<T> gradOutput,
                                           TensorInfo<T> weight, TensorInfo<T> basis,
                                           TensorInfo<int64_t> weightIndex, int n) {
  KERNEL_LOOP(i, n) {
    ptrdiff_t e = i / self.size[1], mIn = i % self.size[1], s, mOut;
    T v = ScalarConvert<int, T>::to(0), b, tmp;
    int64_t wi;
    for (s = 0; s < basis.size[1]; s++) {
      wi = weightIndex.data[e * weightIndex.stride[0] + s * weightIndex.stride[1]];
      b = basis.data[e * basis.stride[0] + s * basis.stride[1]];
      for (mOut = 0; mOut < gradOutput.size[1]; mOut++) {
        tmp = weight.data[wi * weight.stride[0] + mOut * weight.stride[1] + mIn * weight.stride[2]];
        tmp = THCNumerics<T>::mul(tmp, gradOutput.data[e * gradOutput.stride[0] + mOut * gradOutput.stride[1]]);
        tmp = THCNumerics<T>::mul(tmp, b);
        v = THCNumerics<T>::add(v, tmp);
      }
    }
    self.data[e * self.stride[0] + mIn * self.stride[1]] = v;
  }
}

template<typename T>
__global__ void weightingBackwardWeightKernel(TensorInfo<T> self, TensorInfo<T> gradOutput,
                                              TensorInfo<T> src, TensorInfo<T> basis,
                                              TensorInfo<int64_t> weightIndex, int n) {
  KERNEL_LOOP(i, n) {
    ptrdiff_t e = i / gradOutput.size[1], mOut = i % gradOutput.size[1], s, mIn;
    T b, v;
    int64_t wi;
    T g = gradOutput.data[e * gradOutput.stride[0] + mOut * gradOutput.stride[1]];
    for (s = 0; s < weightIndex.size[1]; s++) {
      b = basis.data[e * basis.stride[0] + s * basis.stride[1]];
      wi = weightIndex.data[e * weightIndex.stride[0] + s * weightIndex.stride[1]];
      for (mIn = 0; mIn < src.size[1]; mIn++) {
        v = src.data[e * src.stride[0] + mIn * src.stride[1]];
        v = THCNumerics<T>::mul(THCNumerics<T>::mul(v, b), g);
        atomicAdd(&self.data[wi * self.stride[0] + mIn * self.stride[1] + mOut * self.stride[2]], v);
      }
    }
  }
}

template<typename T>
__global__ void weightingBackwardBasisKernel(TensorInfo<T> self, TensorInfo<T> gradOutput,
                                             TensorInfo<T> src, TensorInfo<T> weight,
                                             TensorInfo<int64_t> weightIndex, int n) {
  KERNEL_LOOP(i, n) {
    ptrdiff_t e = i / gradOutput.size[1], mOut = i % gradOutput.size[1], s, mIn;
    T v, tmp;
    int64_t wi;
    T g = gradOutput.data[e * gradOutput.stride[0] + mOut * gradOutput.stride[1]];
    for (s = 0; s < weightIndex.size[1]; s++) {
      v = ScalarConvert<int, T>::to(0);
      wi = weightIndex.data[e * weightIndex.stride[0] + s * weightIndex.stride[1]];
      for (mIn = 0; mIn < src.size[1]; mIn++) {
        tmp = weight.data[wi * weight.stride[0] + mIn * weight.stride[1] + mOut * weight.stride[2]];
        tmp = THCNumerics<T>::mul(tmp, src.data[e * src.stride[0] + mIn * src.stride[1]]);
        tmp = THCNumerics<T>::mul(tmp, g);
        v = THCNumerics<T>::add(v, tmp);
      }
      atomicAdd(&self.data[e * self.stride[0] + s * self.stride[1]], v);
    }
  }
}

#include "generic/THCWeighting.cu"
#include "THC/THCGenerateFloatTypes.h"
